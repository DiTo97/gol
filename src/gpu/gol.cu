#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Custom includes
#include "gol.h"

/**
 * Initialize all variables and structures required by GoL evolution.
 */
void initialize(life_t *life) {
    // 1. Initialize the random seed
    srand(life->seed);

    // 2. Check if an input file was specified in the args
    // and, in that case, update ncols and nrows.
    //
    // Use defaults, if no file is present.
    FILE *input_ptr = set_grid_dimens_from_file(life);

    // 3. Allocate memory for the grid
    malloc_grid(life);

    // 4. Initialize the grid with DEAD cells
    init_empty_grid(life);

    // 5. Initialize the grid with ALIVE cells...
    if (input_ptr != NULL) { // ...from file, if present...
        init_from_file(life, input_ptr);
    } else {  // ...or randomly, otherwise.
        init_random(life);
    }

    #ifdef GoL_DEBUG
    debug(*life);
    usleep(1000000);
    #endif
}

/**
 * Perform GoL evolution for a given amount of generations on a GPU kernel.
 * 
 * @return tot_gene_time    The total time devolved to GoL evolution
 */
double game(life_t *life) {
    int t;

    struct timeval gstart, gend;
    
    // initializing the whole matrix only if not running with MPI
    initialize(life);

    int ncols = life->ncols;
    int nrows = life->nrows;

    double tot_gene_time = 0.;
    double cur_gene_time = 0.;

    // Check whether the requested block size is valid,
    // or fewer threads have to be assigned to each block
    int block_size = life->block_size <= DEFAULT_MAX_BLOCK_SIZE \
            ? life->block_size : DEFAULT_MAX_BLOCK_SIZE;

    // Check whether the data can be evenly
    // distributed across blocks, or the grid size
    // has to be enlarged accordingly...
    int grid_size = -1;

    if ((ncols*nrows) % block_size == 0)
        grid_size = (int)(ncols*nrows) / block_size;
    else // ...via the formula presented in the HPC course slides
        grid_size = (int)((ncols*nrows + block_size - 1) / block_size);

    // Size of all cells in the grid
    size_t world_size = ncols*nrows * sizeof(bool);
    
    // Init the 1D data structures hosted on GPU
	bool *gpu_grid, *gpu_next_grid;

    // Copy the data from host to device in the 1st grid
    hipMalloc((void **) &gpu_grid, world_size);
    hipMemcpy(gpu_grid, life->grid, world_size,
            hipMemcpyHostToDevice);

    // Fill with DEAD cells the 2nd grid
    hipMalloc((void **) &gpu_next_grid, world_size);
    hipMemset(gpu_next_grid, DEAD, world_size);

	display(*life, false);

    for(t = 0; t < life->timesteps; t++) { 
        // 1. Track the start time
        gettimeofday(&gstart, NULL);
        
		// 2. Evolve the current generation with a CUDA kernel
		evolve<<<grid_size, block_size>>>(
                gpu_grid, gpu_next_grid, nrows, ncols);
        
        // 3. Wait for all CUDA threads to finish
		hipDeviceSynchronize();

        // 4. Swap the memory pointers on GPU
        swap_grids(&gpu_grid, &gpu_next_grid);

        // 5. Track the end time
		gettimeofday(&gend, NULL);

        cur_gene_time = elapsed_wtime(gstart, gend);
        tot_gene_time += cur_gene_time;

        if (is_big(*life)) {
            printf("Generation #%d took %.5f ms\n", t, cur_gene_time);  

            // If the GoL grid is large, print it (to file)
            // only at the end of the last generation
            if (t == life->timesteps - 1) {
				hipMemcpy(life->grid, gpu_grid, world_size,
                        hipMemcpyDeviceToHost);
                display(*life, true);
            }
        } else {
			hipMemcpy(life->grid, gpu_grid, world_size,
                    hipMemcpyDeviceToHost);
            display(*life, true);
        }
	}

    printf("\nEvolved GoL's grid for %d generations - ETA: %.5f ms\n",
            life->timesteps, tot_gene_time);

	// Free the memory on GPU
	hipFree(gpu_grid);
	hipFree(gpu_next_grid);

    return tot_gene_time;
}

/**
 * Perform one evolutionary step of the board, following GoL rules, in this order:
 *     1. A cell is born, if it has exactly 3 neighbours;
 *     2. A cell dies of loneliness, if it has less than 2 neighbours;
 *     3. A cell dies of overcrowding, if it has more than 3 neighbours;
 *     4. A cell survives to the next generation, if it doesn't die of loneliness or overcrowding.
 * 
 * All CUDA threads will access the global memory in order to establish the a cell's next state. Indeed in this case,
 * having each thread to access only 8 different memory locations, we'd argue that memory coalescing, that is,
 * the aggregation step that CUDA already makes under the wood when multiple threads try to access nearby memory location,
 * is an already sufficient optimization step without having to make expicit use of shared memory.
 * 
 * In order to avoid warp divergence with CUDA, here we follow a different approach from that on CPU. Indeed instead of
 * relying on a for loop looking for all neighbours, here we compute their indexes 1-by-1 straight away,
 * as suggested by http://www.marekfiser.com/Projects/Conways-Game-of-Life-on-GPU-using-CUDA
 * 
 * @param gpu_grid         The 1D data on GPU with byte-per-cell density
 * @param gpu_next_grid    The result buffer after evolution on GPU
 * @param ncols            The width of GoL's grid in bytes  (= # of cols)
 * @param nrows            The height of GoL's grid in bytes (= # of rows)
 */
__global__
void evolve(bool *gpu_grid, bool *gpu_next_grid,
        int nrows, int ncols) {
    // # of all cells in the grid
    int world_size = ncols*nrows;

    // 1. Identify the cell the calling thread works on
    // by using this standard CUDA formula
    int cell_id = blockIdx.x*blockDim.x + threadIdx.x;

    // If GoL's data couldn't be evenly distributed across blocks,
    // there will some threads instantiated in the last block
    // without an actual cell to work on; hence, if their target
    // cell Id is outside the world size, they have to return.
    if (cell_id >= world_size)
        return;

    /*
     * A running example with a 3x5 grid, a block size of 4 and a grid size of 4:
     * 0, A, 0, A, 0,
     * 0, 0, 0, A, A,
     * A, 0, 0, 0, 0
     * 
     * with both a block size and a grid size of 4:
     * [0, A, 0, A], [0, 0, 0, 0], [A, A, A, 0] [0, 0, 0, -]
     *  
     * The target cell P is at location (1, 3) = A and its Id is 2*4 + 0 = 8.
     */

    // 2. Retrieve the 0-indexed positions inside the 1D grid of the column, x,
    // and of the start of the row, y, of the target cell
    int x = cell_id % ncols;
    int y = cell_id - x;

    /*
     * x = 8 % 5 = 3
     * y = 8 - 3 = 5
     * 
     * P is indeed in the 3rd column, and the 5th element of
     * the flattened 1D array is the 0 at the start of the 1st row.
     */

    // 3. Retrieve the 0-indexed columns
    // of the left and right neighbours
    int x_left  = (x + ncols - 1) % ncols; 
    int x_right = (x + 1) % ncols;

    /*
     * x_left  = (3 + 5 - 1) % 5 = 2
     * x_right = (3 + 1) % 5 = 4
     */

    // 4. Retrieve the 0-indexed start of the row positions
    // of the upper and lower neighbours
    int y_up   = (y + world_size - ncols) % world_size;
    int y_down = (y + ncols) % world_size;

    /*
     * y_up   = (5 + 15 - 5) % 15 = 0
     * y_down =  (5 + 5) % 15 = 10
     * 
     * Complete the example by yourself as a counterproof
     * that this method works as intended.
     */

    // 5. Count how many neighbours are ALIVE
    int alive_neighbs = gpu_grid[x_left + y_up] // Top-left neighbour
            + gpu_grid[x + y_up]                // Upper neighbour
            + gpu_grid[x_right + y_up]          // Top-right neighbour
            + gpu_grid[x_left + y]              // Left neighbour
            + gpu_grid[x_right + y]             // Right neighbour
            + gpu_grid[x_left + y_down]         // Bottom-left neighbour
            + gpu_grid[x + y_down]              // Lower neighbour
            + gpu_grid[x_right + y_down]        // Bottom-right neighbour

    // 6. Update the next grid with the new state
    gpu_next_grid[x + y] = (alive_neighbs == 3
            || (alive_neighbs == 2 && gpu_grid[x + y]))
            ? ALIVE : DEAD;
}

void cleanup(life_t *life) {  
    free(life->grid);
}

/************************************
 * ================================ *
 ************************************/

int main(int argc, char **argv) {
    struct timeval start, end;

    life_t life;

    gettimeofday(&start, NULL);

    // 1. Initialize vars from args
    parse_args(&life, argc, argv);

    FILE *input_ptr = set_grid_dimens_from_file(&life);

	// 2. Launch the simulation
    double cum_gene_time = game(&life);
    cleanup(&life);

    gettimeofday(&end, NULL);

    double elapsed_prog_wtime = elapsed_wtime(start, end);
    
    #ifdef GoL_LOG
    FILE *log_ptr = init_log_file(life, 1);

    log_data(log_ptr, life.timesteps, cum_gene_time,
            elapsed_prog_wtime);

    fflush(log_ptr);
    fclose(log_ptr);
    #endif

    printf("Finalized the program - ETA: %.5f ms\n\n", elapsed_prog_wtime);
}
