#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "gol.h"
#include "utils.h"


//TODO: add display function which makes use of CUDA

/**
 * Print the current GoL board to console.
 */
 void show(struct life_t life) {
    int x, y;

    int ncols = life.num_cols;
    int nrows = life.num_rows;

    // \033[H: Move cursor to top-left corner;
    // \033[J: Clear console.
    printf("\033[H\033[J");

    for (x = 0; x < nrows; x++) {
        for (y = 0; y < ncols; y++)
            printf(life.grid[x*ncols+y] == ALIVE ? "\033[07m  \033[m" : "  ");

        printf("\033[E");
    }

    fflush(stdout);
    usleep(160000);
}

/**
 * Print the current GoL board to either console or file depending on whether its size is larger than DEFAULT_MAX_SIZE.
 */
void display(struct life_t life, bool append) {
    if(is_big(life)) printbig(life, append);
    else show(life);
}

/***********************
 * Evolution functions *
 ***********************/

/**
 * Initialize all variables and structures required by GoL evolution.
 */
void initialize(struct life_t *life) {
    // 1. Initialize the random seed
    srand(life->seed);

    // 2. Check if an input file was specified in the args
    // and, in that case, update num_cols and num_rows.
    //
    // Use defaults, if no file is present.
    FILE *input_ptr = set_grid_dimens_from_file(life);

    // 3. Allocate memory for the grid
    malloc_grid(life);

    // 4. Initialize the grid with DEAD cells
    init_empty_grid(life);

    // 5. Initialize the grid with ALIVE cells...
    if (input_ptr != NULL) { // ...from file, if present...
        init_from_file(life, input_ptr);
    } else {  // ...or randomly, otherwise.
        init_random(life);
    }

    #ifdef GoL_DEBUG
    debug(*life);
    usleep(1000000);
    #endif
}


// CUDA facts:
//
// On devices of compute capability 2.x and beyond, 32-bit integer multiplication is natively supported,
// but 24-bit integer multiplication is not. __[u]mul24 is therefore implemented using multiple instructions
// and should not be used.
//
// Integer division and modulo operation are costly: below 20 instructions on devices of compute capability 2.x and
// higher. They can be replaced with bitwise operations in some cases: If n is a power of 2, (i/n) is equivalent to
// (i>>log2(n)) and (i%n) is equivalent to (i&(n-1)); the compiler will perform these conversions if n is literal.

/**
 * Perform one evolutionary step of the board, following GoL rules, in this order:
 * 
 *     1. A cell is born, if it has exactly 3 neighbours;
 *     2. A cell dies of loneliness, if it has less than 2 neighbours;
 *     3. A cell dies of overcrowding, if it has more than 3 neighbours;
 *     4. A cell survives to the next generation, if it does not die of loneliness or overcrowding.
 */
///
/// @param curr_grid  Linearized 2D array of life data with byte-per-cell density.
/// @param num_cols  Width of life world in cells (bytes).
/// @param num_rows  Height of life world in cells (bytes).
/// @param next_grid  Result buffer in the same format as input.
__global__ void cuda_evolve(bool * curr_grid, bool * next_grid, int num_rows, int num_cols) {
    // Total number of cells in the grid
	int world_size = num_cols * num_rows;

    /* for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x;
        cellId < world_size;
        cellId += blockDim.x * gridDim.x) { */
    uint cellId = blockIdx.x * blockDim.x + threadIdx.x;
    // retrieve index of the column associated with cellId
    if (cellId >= world_size) {
        return;
    }
    uint x = cellId % num_cols;
    // retrieve the absolute index, inside the 1D board, of the start of the row which corresponds to cellId
    // absolute: the modulo operator is not applied (e.g. with 4 blocks, 4 threads per block, it would yield 15)
    uint yAbs = cellId - x;

    // retrieve the indexes of the columns of the left and right neibors, respectively
    // for example, if each block is composed by 4 threads, then the possible values for 
    // xLeft and xRight are integers in [0, 3]
    uint xLeft = (x + num_cols - 1) % num_cols; 
    uint xRight = (x + 1) % num_cols;

    // retrieve the absolute index, in the 1D board, of the start of the row which corresponds to the upper neighbor
    uint yAbsUp = (yAbs + world_size - num_cols) % world_size;
    // retrieve the absolute index, in the 1D board, of the start of the row which corresponds to the lower neighbor
    uint yAbsDown = (yAbs + num_cols) % world_size;

    // Count alive neighbors. To do so, use the indexes computed in the previous steps. 
    uint alive_cells = curr_grid[xLeft + yAbsUp] + curr_grid[x + yAbsUp] + curr_grid[xRight + yAbsUp] \
        + curr_grid[xLeft + yAbs] + curr_grid[xRight + yAbs] \
        + curr_grid[xLeft + yAbsDown] + curr_grid[x + yAbsDown] + curr_grid[xRight + yAbsDown];

    // ternary operator used to improve performance and avoid warp divergence
    // x + yAbs represents the index of the current cell processed by a thread on the 1D board
    next_grid[x + yAbs] = alive_cells == 3 || (alive_cells == 2 && curr_grid[x + yAbs]) ? 1 : 0;
    //}
}

/**
 * Perform GoL evolution for a given amount of generations and measure execution times.
*/
// TODO: implement CUDA printf for GoL board
double game(struct life_t *life) {
    int t;

    struct timeval start, end;
    
    // initializing the whole matrix only if not running with MPI
    initialize(life);

    int ncols = life->num_cols;
    int nrows = life->num_rows;

    double cum_gen_time = 0.;
    double cur_time = 0.;
	
	// compute number of threads, check whether this number does not exceed the max number of threads per block
	int threads_count = life->nthreads_per_block <= DEFAULT_MAX_NUM_THREADS_PER_BLOCK ? life->nthreads_per_block : DEFAULT_MAX_NUM_THREADS_PER_BLOCK;
	int blocks_count;
	if ((ncols*nrows)%threads_count!=0) {
		// if the number of threads per block is not a multiple of the board size, compute the number of blocks accordingly
		blocks_count = (int)((ncols * nrows + (threads_count - 1))/threads_count);
	}
	else {
		// else, compute the number of threads by diving the total size of the board
		blocks_count = (ncols * nrows) / threads_count;
	}
	
	bool* curr, *next;
	// allocate memory on the device, the board is again expressed as a single array
	hipMalloc((void **)&curr, ncols * nrows * sizeof(bool));
	// copy board to device, on the allocated space
	hipMemcpy(curr, life->grid, ncols * nrows * sizeof(bool), hipMemcpyHostToDevice);

	// initialize another board on the GPU with zeros
	hipMalloc((void **)&next, ncols * nrows * sizeof(bool));
	hipMemset(next, 0, ncols * nrows * sizeof(bool));

	display(*life, false);

	// each block is associated with threads_count threads
	dim3 block_size(threads_count);
	// the grid contains block_count blocks
	dim3 grid_size(blocks_count);

    for(t = 0; t < life->timesteps; t++) { 
        // 1. Track the start time
        gettimeofday(&start, NULL);
        
		// 2. Let the current generation evolve, using a CUDA kernel
		cuda_evolve<<<grid_size, block_size>>>(curr, next, life->num_rows, life->num_cols);
        
		// TODO: implement ifdefs to remove synchronization across timesteps, in case of "performance mode"
		hipDeviceSynchronize();

        // 3. Track the end time
		gettimeofday(&end, NULL);
        

		bool* temp = curr;
		curr=next;
		next=temp;

        cur_time = elapsed_wtime(start, end);
        cum_gen_time += cur_time;

        if (is_big(*life)) {
            printf("Generation #%d took %.5f ms\n", t, cur_time);  

            // If the GoL grid is large, print it (to file)
            // only at the end of the last generation
            if (t == life->timesteps - 1) {
				hipMemcpy(life->grid, curr, ncols * nrows * sizeof(bool), hipMemcpyDeviceToHost);
                display(*life, true);
            }
        } else {
			hipMemcpy(life->grid, curr, ncols * nrows * sizeof(bool), hipMemcpyDeviceToHost);
            display(*life, true);
        }

	}
	
    printf("\nTotal processing time of GoL evolution for %d generations: %.5f ms\n",
        life->timesteps, cum_gen_time);

	// free memory on GPU
	hipFree(curr);
	hipFree(next);

    return cum_gen_time;
}

void cleanup(struct life_t *life) {  
    free(life->grid);
    free(life->next_grid);
}

/************************************
 * ================================ *
 ************************************/

int main(int argc, char **argv) {

    struct life_t life;
    struct timeval start, end;

    gettimeofday(&start, NULL);

    // 1. Initialize vars from args
    parse_args(&life, argc, argv);

    // reading the file if present and setting life dimensions
    FILE *input_ptr = set_grid_dimens_from_file(&life);

	// 2. Launch the simulation
    double cum_gen_time = game(&life);

    // 3. Free the memory
    cleanup(&life);

    gettimeofday(&end, NULL);

    double elapsed_prog_wtime = elapsed_wtime(start, end);
    
    printf("The total execution time is %.5f ms", elapsed_prog_wtime);

    #ifdef GoL_DEBUG
    FILE *log_ptr = init_log_file(life);
    log_data(log_ptr, life.timesteps, cum_gen_time, elapsed_prog_wtime);
    fflush(log_ptr);
    fclose(log_ptr);
    #endif
}